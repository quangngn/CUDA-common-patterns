#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_util.cu"

int main()
{
    clock_t start, end;

    // init size
    int size = 1 << 24;
    size_t byte_size = size * sizeof(int);

    int *arr;
    arr = (int *)malloc(byte_size);
    init_array_cpu(arr, size, RAND_INIT);

    int *d_arr;

    // time hipMalloc
    start = clock();
    cudaErrorCk(hipMalloc((int **)&d_arr, byte_size));
    end = clock();
    printf("Cuda malloc takes %ld\n", end - start);

    // time hipMemcpy
    start = clock();
    cudaErrorCk(hipMemcpy(d_arr, arr, byte_size, hipMemcpyHostToDevice));
    end = clock();
    printf("Cuda memcpy host to device takes %ld\n", end - start);

    // time hipMemcpy
    start = clock();
    cudaErrorCk(hipMemcpy(arr, d_arr, byte_size, hipMemcpyDeviceToHost));
    end = clock();
    printf("Cuda memcpy device to host takes %ld\n", end - start);

    // what about malloc pinned mem
    int *pinned_arr;
    start = clock();
    cudaErrorCk(hipHostMalloc((int **)&pinned_arr, byte_size));
    end = clock();
    printf("Cuda malloc pinned takes %ld\n", end - start);
}